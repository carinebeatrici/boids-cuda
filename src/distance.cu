#include "hip/hip_runtime.h"
// Calculo das distancias e forças entre as particulas dentro da GPU

#include <math.h>
//#include <unistd.h>
#include <hip/hip_runtime.h>
//#include <hipfft/hipfft.h>
#include <math.h>

#include "param.h"
#include "particle.h"

	 

__global__ void G_distance(particle *G_boid, float *G_Fx, float *G_Fy ,float *G_sum_Vx, float *G_sum_Vy)
{
   //---------------locais--------------------------------------------------
   int i,j;
   float dx,dy,L_TENTATIVAh=L_TENTATIVA*0.5f,distance;
   float r_max2=R_MAX*R_MAX;
   float auxiliar;
   //O loop inplicito vai ser na variavel i
   //Cada thread vai calcular as forcas para uma particula
   i = (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
   //------------------todo mundo com todo mundo----------------------------
   //------------------dentro da memoria global-----------------------------
   G_Fx[i]=0.0f;
   G_Fy[i]=0.0f;
   G_sum_Vx[i]=0.0f;
   G_sum_Vy[i]=0.0f;
   for(j=0;j<N;j++)
     {     
	if(i!=j)
	  {
	     dx=G_boid[j].x-G_boid[i].x;
	     dy=G_boid[j].y-G_boid[i].y;
	     if(dx>L_TENTATIVAh)dx=dx-L_TENTATIVA;else if(dx<-L_TENTATIVAh) dx=L_TENTATIVA+dx;
	     if(dy>L_TENTATIVAh)dy=dy-L_TENTATIVA;else if(dy<-L_TENTATIVAh) dy=L_TENTATIVA+dy;
	     distance=dx*dx+dy*dy;
	     //check if particles are interacting
	     if(distance<r_max2)
	       {
		  //align
		  if(j<N1&&i<N1)
		    {
		       G_sum_Vx[i]+= ALPHA11 * G_boid[j].vx;
		       G_sum_Vy[i]+= ALPHA11 * G_boid[j].vy;
		    }
		  else
		    if(j>N1&&i>N1)
		      {
			 G_sum_Vx[i]+= ALPHA22 * G_boid[j].vx;
			 G_sum_Vy[i]+= ALPHA22 * G_boid[j].vy;
		      }
		  else
		    {
		       G_sum_Vx[i]+= ALPHA12 * G_boid[j].vx;
		       G_sum_Vy[i]+= ALPHA12 * G_boid[j].vy;
		    }
		    distance=sqrt(distance);
		  //hard core replusion
		  if (distance<=R_CORE) 
		    { //hard core replusion
		       auxiliar=-FORCA_CORE/distance;
		       G_Fx[i]+=dx*auxiliar;
		       G_Fy[i]+=dy*auxiliar;
		    }
		  else
		    {
		       //valid force range
		       if(j<N1&&i<N1)
			 {
			    G_Fx[i]-=BETA11*dx*(1.0f/distance-FORCA_CORE);
			    G_Fy[i]-=BETA11*dy*(1.0f/distance-FORCA_CORE);
			 }
		       else
			 {
			    if(i>=N1&&j>=N1)
			      {
				 G_Fx[i]-=BETA22*dx*(1.0f/distance-FORCA_CORE);
				 G_Fy[i]-=BETA22*dy*(1.0f/distance-FORCA_CORE);
			      }
			    else
			      {
				 if((i>=N1&&j<N1)||(i<N1&&j>=N1))
				   {
				      G_Fx[i]-=BETA12*dx*(1.0f/distance-FORCA_CORE);
				      G_Fy[i]-=BETA12*dy*(1.0f/distance-FORCA_CORE);
				   }	 
			      }  
			 }
		    }
	       }
	  }
     } 
};


void calculate_distance(particle *boid , float *fx, float *fy, float *sum_vx, float *sum_vy)
{
   int i,j;
   float forca;
   float dx,dy;
   float distance;
   // type 1 and type 1
   for(i=0;i<N1;i++)
     {
	fx[i]=0.00f;
	fy[i]=0.00f;
	sum_vx[i]=0.00f;
	sum_vy[i]=0.00f;
	for(j=0;j<N1;j++)
	  {
	     dx=boid[i].x-boid[j].x;
	     dy=boid[i].y-boid[j].y;
	     if(dx>L_TENTATIVA/2.0f)dx=dx-L_TENTATIVA;
	     if(dy>L_TENTATIVA/2.0f)dy=dy-L_TENTATIVA;
	     if(dx<-L_TENTATIVA/2.0f)dx=dx+L_TENTATIVA;
	     if(dy<-L_TENTATIVA/2.0f)dy=dy+L_TENTATIVA;
	     distance=sqrt((dx*dx)+(dy*dy));
	     if(distance<R_CORE)
	       {
		  forca = FORCA_CORE; 
		  if(distance>0.00010f)
		    {
		       sum_vx[i]+= ALPHA11 * boid[j].vx;
		       sum_vy[i]+= ALPHA11 * boid[j].vy;
		    }		  
	       }
	     else 
	       {
		  if(distance<R_MAX)
		    {
		       forca = BETA11 * (1.00f - distance/R_EQ);
		       sum_vx[i]+= ALPHA11 * boid[j].vx;
		       sum_vy[i]+= ALPHA11 * boid[j].vy;
		    }
		  else
		    {
		       forca = 0.00f;
		    }
	       }
	     if(distance>0.00010f)
	       {
		  fx[i] += (forca * dx) / distance;
		  fy[i] += (forca * dy) / distance;
	       }
	  }
     }
   // type 1 and type 2
   for(i=0;i<N1;i++)
     {
	for(j=N1;j<N;j++)
	  {
	     dx=boid[i].x-boid[j].x;
	     dy=boid[i].y-boid[j].y;
	     if(dx>L_TENTATIVA/2.0f)dx=dx-L_TENTATIVA;
	     if(dy>L_TENTATIVA/2.0f)dy=dy-L_TENTATIVA;
	     if(dx<-L_TENTATIVA/2.0f)dx=dx+L_TENTATIVA;
	     if(dy<-L_TENTATIVA/2.0f)dy=dy+L_TENTATIVA;
	     distance=sqrt((dx*dx)+(dy*dy));
	     if(distance<R_CORE)
	       {
		  forca = FORCA_CORE; 
		  if(distance>0.00010f)
		    {
		       sum_vx[i]+= ALPHA12 * boid[j].vx;
		       sum_vy[i]+= ALPHA12 * boid[j].vy;
		    }		  
	       }
	     else 
	       {
		  if(distance<R_MAX)
		    {
		       forca = BETA12 * (1.00f - distance/R_EQ);
		       sum_vx[i]+= ALPHA12 * boid[j].vx;
		       sum_vy[i]+= ALPHA12 * boid[j].vy;
		    }
		  else
		    {
		       forca = 0.00f;
		    }
	       }
	     if(distance>0.00010f)
	       {
		  fx[i] += (forca * dx) / distance;
		  fy[i] += (forca * dy) / distance;
	       }
	  }
     }
   // type 2 and type 1
   for(i=N1;i<N;i++)
     {
	fx[i]=0.00f;
	fy[i]=0.00f;
	sum_vx[i]=0.00f;
	sum_vy[i]=0.00f;
	for(j=0;j<N1;j++)
	  {
	     dx=boid[i].x-boid[j].x;
	     dy=boid[i].y-boid[j].y;
	     if(dx>L_TENTATIVA/2.0f)dx=dx-L_TENTATIVA;
	     if(dy>L_TENTATIVA/2.0f)dy=dy-L_TENTATIVA;
	     if(dx<-L_TENTATIVA/2.0f)dx=dx+L_TENTATIVA;
	     if(dy<-L_TENTATIVA/2.0f)dy=dy+L_TENTATIVA;
	     distance=sqrt((dx*dx)+(dy*dy));
	     if(distance<R_CORE)
	       {
		  forca = FORCA_CORE; 
		  if(distance>0.00010f)
		    {
		       sum_vx[i]+= ALPHA12 * boid[j].vx;
		       sum_vy[i]+= ALPHA12 * boid[j].vy;
		    }		  
	       }
	     else 
	       {
		  if(distance<R_MAX)
		    {
		       forca = BETA12 * (1.00f - distance/R_EQ);
		       sum_vx[i]+= ALPHA12 * boid[j].vx;
		       sum_vy[i]+= ALPHA12 * boid[j].vy;
		    }
		  else
		    {
		       forca = 0.00f;
		    }
	       }
	     if(distance>0.00010f)
	       {
		  fx[i] += (forca * dx) / distance;
		  fy[i] += (forca * dy) / distance;
	       }
	  }
     }
   // type 2 and type 2 
   for(i=N1;i<N;i++)
     {
	for(j=N1;j<N;j++)
	  {
	     dx=boid[i].x-boid[j].x;
	     dy=boid[i].y-boid[j].y;
	     if(dx>L_TENTATIVA/2.0f)dx=dx-L_TENTATIVA;
	     if(dy>L_TENTATIVA/2.0f)dy=dy-L_TENTATIVA;
	     if(dx<-L_TENTATIVA/2.0f)dx=dx+L_TENTATIVA;
	     if(dy<-L_TENTATIVA/2.0f)dy=dy+L_TENTATIVA;
	     distance=sqrt((dx*dx)+(dy*dy));
	     if(distance<R_CORE)
	       {
		  forca = FORCA_CORE; 
		  if(distance>0.00010f)
		    {
		       sum_vx[i]+= ALPHA22 * boid[j].vx;
		       sum_vy[i]+= ALPHA22 * boid[j].vy;
		    }		  
	       }
	     else 
	       {
		  if(distance<R_MAX)
		    {
		       forca = BETA22 * (1.00f - distance/R_EQ);
		       sum_vx[i]+= ALPHA22 * boid[j].vx;
		       sum_vy[i]+= ALPHA22 * boid[j].vy;
		    }
		  else
		    {
		       forca = 0.00f;
		    }
	       }
	     if(distance>0.00010f)
	       {
		  fx[i] += (forca * dx) / distance;
		  fy[i] += (forca * dy) / distance;
	       }
	  }
     }
}
