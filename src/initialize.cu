#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "param.h"
#include "particle.h"
#include "cuda-param.h"

#define TWO_PI 6.283185f

__global__ void G_initialize_random_generator(unsigned int seed, hiprandState_t *state)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(seed, idx, 0, &state[idx]);
};

__global__ void G_initialize(particle *G_boid,  hiprandState_t *G_random_state, unsigned int *G_random_number)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  float angle;
  float auxiliar_rand;
  //inicializacao das posicoes e velocidades das particulas
  if(idx<N1)
    {
      angle = hiprand_uniform(&G_random_state[idx]);
      angle = TWO_PI * angle;
      G_boid[idx].vx = V1 * cos(angle);
      G_boid[idx].vy = V1 * sin(angle);
      auxiliar_rand = hiprand_uniform(&G_random_state[idx]);
      G_boid[idx].x  = L_TENTATIVA/3.0f * auxiliar_rand + L_TENTATIVA/2.0f;
      auxiliar_rand = hiprand_uniform(&G_random_state[idx]);
      G_boid[idx].y  = L_TENTATIVA/3.0f * auxiliar_rand + L_TENTATIVA/2.0f;
      G_boid[idx].label = idx;
      G_boid[idx].v0=V1;
    }
    else
    if(idx<N)
    {
      angle = hiprand_uniform(&G_random_state[idx]);
      angle = TWO_PI * angle;
      G_boid[idx].vx = V2 * cos(angle);
      G_boid[idx].vy = V2 * sin(angle);
      auxiliar_rand = hiprand_uniform(&G_random_state[idx]);
      G_boid[idx].x  = L_TENTATIVA/3.0f * auxiliar_rand + L_TENTATIVA/2.0f;
      auxiliar_rand = hiprand_uniform(&G_random_state[idx]);
      G_boid[idx].y  = L_TENTATIVA/3.0f * auxiliar_rand + L_TENTATIVA/2.0f;
      G_boid[idx].label = idx;
      G_boid[idx].v0=V2;
    }
};



void inicializa(particle *boid, int *random_seed)
{
   int i;   
   float angle; 
   for(i=0;i<N1;i++)
     {
	angle = drand48();
	angle *= TWO_PI;
	boid[i].vx = V1 * cos(angle);
	boid[i].vy = V1 * sin(angle);
	boid[i].x  = L_TENTATIVA/3.0f * drand48() + L_TENTATIVA/2.0f;
	boid[i].y  = L_TENTATIVA/3.0f * drand48() + L_TENTATIVA/2.0f;
	boid[i].label = i;
	boid[i].v0 = V1;
     }
   for(i=N1;i<N;i++)
     {
	angle = drand48();
	angle *= TWO_PI;
	boid[i].vx = V2 * cos(angle);
	boid[i].vy = V2 * sin(angle);
	boid[i].x  = L_TENTATIVA/3.0f * drand48() + L_TENTATIVA/2.0f;
	boid[i].y  = L_TENTATIVA/3.0f * drand48() + L_TENTATIVA/2.0f;
	boid[i].label = i;
	boid[i].v0 = V2;
     }
}
