#include "hip/hip_runtime.h"
/* Modelo de Vicsek 
 * Inicio: 28/07/2016
 * Versão CUDA - 08/09/2016
 * Sem caixas na memoria global
 */ 

#define TWO_PI 6.2830f

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "param.h"
#include "particle.h"
#include "cuda-param.h"

#include "distance.cuh"
#include "update.cuh"
#include "save_position.cuh"
 

// Cabeçalhos das funções em C (CPU)
float  *Alocate_vector_real (int vector_size);
float  *Free_vector_real (int vector_size, float *vector);
void   save_position(FILE *arquivo, particle *boid, int time_step);

//void  inicializa(particle *boid, int *random_seed);
//void  calculate_distance(particle *boid, float *fx, float *fy, float *sum_vx, float *sum_vy);
//void  update_position(particle *boid, float *fx, float *fy, float *sum_vx, float *sum_vy);

//Cabeçalh das funçõ em CUDA (GPU)
__global__ void G_initialize(particle *G_boid,  hiprandState_t *G_random_state, unsigned int *G_random_number);
__global__ void G_initialize_random_generator(unsigned int seed, hiprandState_t *state);
//__global__ void G_update_position(particle *G_boid, float *G_Fx, float *G_Fy ,float *G_sum_Vx, float *G_sum_Vy, hiprandState_t *state, unsigned int *G_random_number);
//__global__ void G_calculate_distance(particle *G_boid, float *G_Fx, float *G_Fy ,float *G_sum_Vx, float *G_sum_Vy);



int main (void)
{
//   printf("Começo do programa  \n");
//   float *fx,*fy,*sum_vx,*sum_vy; // coordinates, velocities and forces projections
   float exec_time_gpu=0.0,exec_time_gpu_total=0.0,exec_time_cpu=0.0;
   float exec_time_total=0.0;
   clock_t time_cpu_init,time_cpu_end;
   hipEvent_t start,stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   time_cpu_init=clock();
   float *G_fx,*G_fy,*G_sum_vx,*G_sum_vy; // coordinates, velocities and forces projections on GPU
   particle *boid;   //particulas em CPU para salvar os arquivos
   particle *G_boid; //particulas em GPU para calculos
   hiprandState_t *G_random_state; //sementes aleatorias
   unsigned int *G_random_number;  // vetores com numeros aleatorios em GPU
   long int time_step=0;
   FILE *arquivo_saida;
   arquivo_saida=fopen("data/posicoes.dat","w");
   // Allocating stuff   
//   printf(" Alocando coisas...................................");
   boid = (particle*) malloc (N * sizeof(particle)); // Alocando vetor de estruturas em CPU
   hipMalloc((void**) &G_boid, (N * sizeof(particle))); //N random numbers each time step
   hipMalloc((void**) &G_random_state, N * sizeof(hiprandState_t)); //N random numbers each time step
   hipMalloc((void**) &G_random_number, N * sizeof(unsigned int)); //N random numbers each time step   
   hipMalloc((void**) &G_fx, N * sizeof(float)); 
   hipMalloc((void**) &G_fy, N * sizeof(float)); 
   hipMalloc((void**) &G_sum_vx, N * sizeof(float));
   hipMalloc((void**) &G_sum_vy, N * sizeof(float));
//   printf("OK  \n");
//   fx     = Alocate_vector_real (N);
//   fy     = Alocate_vector_real (N);
//   sum_vx = Alocate_vector_real (N);
//   sum_vy = Alocate_vector_real (N);

//   inicializa(boid);
//   printf(" Antes inicializar random generator................");
   time_cpu_end = clock();
   exec_time_cpu = time_cpu_end-time_cpu_init;
   time_cpu_init=clock();
   hipEventRecord(start,0);
   G_initialize_random_generator<<<BLOCKS,THREADS>>>(time(0),G_random_state);
//   printf("OK  \n");
// Inicia sementes aleatorias em GPU
//   printf("Inicializando boids................................");
   G_initialize<<<BLOCKS,THREADS>>>(G_boid,G_random_state,G_random_number);
//   printf("OK  \n");
//   printf("Copiando dados.....................................");
   hipMemcpy(boid,G_boid,(N*sizeof(particle)),hipMemcpyDeviceToHost);
   hipEventRecord(stop,0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&exec_time_gpu,start,stop);
   exec_time_gpu_total+=exec_time_gpu;
   time_cpu_init=clock();
//   printf("OK  \n");
//   printf("x: %f y: %f vx: %f vy: %f  \n",boid[0].x,boid[0].y,boid[0].vx,boid[0].vy);
//   printf("x: %f y: %f vx: %f vy: %f  \n",boid[N-1].x,boid[N-1].y,boid[N-1].vx,boid[N-1].vy);
//   printf("Salvando Particulas................................");
   save_position(arquivo_saida, boid, time_step);
//   printf("OK  \n");
//   printf("Stuff alocated \n");
   // Time loop
   while (time_step < TIME_FINAL)
     {
//	printf("Loop time...........................................,%d  \n",time_step);
	time_step++;
//	calculate_distance(boid,fx,fy,sum_vx,sum_vy);
	time_cpu_end=clock();
	exec_time_cpu += time_cpu_end-time_cpu_init;
	hipEventRecord(start,0);
        G_calculate_distance<<<BLOCKS,THREADS>>>(G_boid,G_fx,G_fy,G_sum_vx,G_sum_vy);
//	update_position(boid,fx,fy,sum_vx,sum_vy);
        G_update_position<<<BLOCKS,THREADS>>>(G_boid,G_fx,G_fy,G_sum_vx,G_sum_vy,G_random_state,G_random_number);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&exec_time_gpu,start,stop); 
	exec_time_gpu_total+=exec_time_gpu;
	time_cpu_init=clock();
	if(time_step%1000==0)
	  {
	    hipMemcpy(boid,G_boid,N*sizeof(particle),hipMemcpyDeviceToHost);
	    save_position(arquivo_saida, boid, time_step);
	  }
     }
   // LIBERATING MEMORY
//   printf("Stuff dealocated \n");
//   fx = Free_vector_real (N, fx);
//   fy = Free_vector_real (N, fy);
//   sum_vx = Free_vector_real (N, sum_vx);
//   sum_vy = Free_vector_real (N, sum_vy);
   hipFree(G_boid);
   hipFree(G_random_state);
   hipFree(G_random_number);
   hipFree(G_fx);
   hipFree(G_fy);
   hipFree(G_sum_vx);
   hipFree(G_sum_vy);
   fclose(arquivo_saida);
   time_cpu_end=clock();
   exec_time_cpu += time_cpu_end-time_cpu_init;
   exec_time_cpu /= (float)CLOCKS_PER_SEC;
   exec_time_gpu_total /= 1000.0;
   exec_time_total= exec_time_cpu+ exec_time_gpu_total;
   printf("   \n");
   printf("tempo total: %f s numero de particulas: %d \n tempo de CPU: %f s tempo de GPU: %f s\n",
	  exec_time_total, N, exec_time_cpu, exec_time_gpu_total );
   printf("Threads: %d Blocks: %d \n",THREADS,BLOCKS);
//   printf("Num of loads: %d \n",NUM_LOADS);
   return 0;
};




/*
void  save_position(FILE *arquivo, particle *boid, int time_step)
{
   int i;
//   printf("entrou");
   fprintf(arquivo,"%d \n",time_step);
   for(i=0;i<N;i++)fprintf(arquivo,"%f %f %f %f\n",boid[i].x,boid[i].y,boid[i].vx,boid[i].vy);
};
*/

/*
__global__ void G_calculate_distance(particle *G_boid, float *G_Fx, float *G_Fy ,float *G_sum_Vx, float *G_sum_Vy)
{
   //---------------locais--------------------------------------------------
   int i,j;
   float dx,dy,L_TENTATIVAh=L_TENTATIVA*0.5f,distance;
   float r_max2=R_MAX*R_MAX;
   float auxiliar;
   //O loop inplicito vai ser na variavel i
   //Cada thread vai calcular as forcas para uma particula
   i = (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
   //------------------todo mundo com todo  mundo----------------------------
   //------------------dentro da memoria global-----------------------------
   if(i<N)
     {
	G_Fx[i]=0.0f;
	G_Fy[i]=0.0f;
	G_sum_Vx[i]=0.0f;
	G_sum_Vy[i]=0.0f;
	for(j=0;j<N;j++)
	  {
	     if(i!=j)
	       {
		  dx=G_boid[i].x-G_boid[j].x;
		  dy=G_boid[i].y-G_boid[j].y;
		  if(dx>L_TENTATIVAh)dx=dx-L_TENTATIVA;else if(dx<-L_TENTATIVAh) dx=L_TENTATIVA+dx;
		  if(dy>L_TENTATIVAh)dy=dy-L_TENTATIVA;else if(dy<-L_TENTATIVAh) dy=L_TENTATIVA+dy;
		  distance=dx*dx+dy*dy;
		  //check if particles are interacting
		  if(distance<r_max2)
		    {
		       //align
		       if(j<N1&&i<N1)
			 {
			    G_sum_Vx[i]+= ALPHA11 * G_boid[j].vx;
			    G_sum_Vy[i]+= ALPHA11 * G_boid[j].vy;
			 }
		       else
			 if(j>N1&&i>N1)
			   {
			      G_sum_Vx[i]+= ALPHA22 * G_boid[j].vx;
			      G_sum_Vy[i]+= ALPHA22 * G_boid[j].vy;
			   }
		       else
			 {
			    G_sum_Vx[i]+= ALPHA12 * G_boid[j].vx;
			    G_sum_Vy[i]+= ALPHA12 * G_boid[j].vy;
			 }
		       distance=sqrt(distance);
		       //hard core replusion
		       if (distance<=R_CORE)
			 { //hard core replusion
			    auxiliar=FORCA_CORE/distance;
			    G_Fx[i]+=dx*auxiliar;
			    G_Fy[i]+=dy*auxiliar;
			 }
		       else
			 {
			    //valid force range
			    if(j<N1&&i<N1)
			      {
				 G_Fx[i]+=BETA11*dx*(1.0f-distance/R_EQ);
				 G_Fy[i]+=BETA11*dy*(1.0f-distance/R_EQ);
			      }
			    else
			      {
				 if(i>=N1&&j>=N1)
				   {
				      G_Fx[i]+=BETA22*dx*(1.0f-distance/R_EQ);
				      G_Fy[i]+=BETA22*dy*(1.0f-distance/R_EQ);
				   }
				 else
				   {
				      if((i>=N1&&j<N1)||(i<N1&&j>=N1))
					{
					   G_Fx[i]+=BETA12*dx*(1.0f-distance/R_EQ);
					   G_Fy[i]+=BETA12*dy*(1.0f-distance/R_EQ);
					}
				   }
			      }
			 }
		    }
	       }
	  }
     }
};
*/   

/*
__global__ void G_update_position(particle *G_boid, float *G_Fx, float *G_Fy ,float *G_sum_Vx, 
				  float *G_sum_Vy, hiprandState_t *state, unsigned int * G_random_number)
{
   int i=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
   float module_displace;
   float component_x,component_y;
   float angle;
   if (i<N)
     {
	G_random_number[i] = hiprand(state) % RAND_MAX;
	angle = TWO_PI * (float) G_random_number[i] /RAND_MAX;
	component_x =  ( G_sum_Vx[i] + G_Fx[i] + ETA * cos(angle));
	component_y =  ( G_sum_Vy[i] + G_Fy[i] + ETA * sin(angle));
	module_displace = sqrt((component_x*component_x)+(component_y*component_y));
	G_boid[i].x = G_boid[i].x + G_boid[i].v0 * component_x/module_displace;
	G_boid[i].y = G_boid[i].y + G_boid[i].v0 * component_y/module_displace;
	//boundary conditions
	if(G_boid[i].x>=L_TENTATIVA)G_boid[i].x=G_boid[i].x-L_TENTATIVA;
	if(G_boid[i].y>=L_TENTATIVA)G_boid[i].y=G_boid[i].y-L_TENTATIVA;
	if(G_boid[i].x<0)G_boid[i].x=G_boid[i].x+L_TENTATIVA;
	if(G_boid[i].y<0)G_boid[i].y=G_boid[i].y+L_TENTATIVA;
	G_boid[i].vx = G_boid[i].v0 * component_x/module_displace;
	G_boid[i].vy = G_boid[i].v0 * component_y/module_displace;
     }
};
  */ 