#include "hip/hip_runtime.h"
/*
 *  tentativas para otimizacao utilizando a memoria compartilhada
 *  calcuco do gama feito ok mas nao ficou mais rapido :(
 *  Ficou mais lento precisamos otimizar mesmo é a update
 *  Ficou mais rápido ajustando o número de blocos e threads
 *  ~2.4x mais rapido de 5.91s para 2.5s
 */ 
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "paramfile.h"
#include "fileutils.h"
#include "stringutils.h"

#include "particles.h"

using namespace std;

#define N_tot 6000
//===========================================================================
//===========================================================================
//===========================================================================
// CUDA  kernel&device functions
//---------------------------------------------------------------------------

__global__ void G_setrconst(int N,REAL *a,REAL c) {
    int idx=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(idx<N) {a[idx]=c;}
};

__global__ void G_seticonst(int N,int *a,int c) {
    int idx=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(idx<N) {a[idx]=c;}
};

//---------------------------------------------------------------------------
__device__ int Gran0(int rs)
{
	int k;
	k=rs/IQ0;
	rs=IA0*(rs-k*IQ0)-IR0*k;
	if (rs < 0) rs += IM0;
	return rs;
};

__global__ void G_addrand(int N,REAL *A,int *rs,REAL T)
{
//calculo dos numeros aleatorios 
        int i=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	int rn;
	if(i<N)
	{
		rn=rs[i];
		rn=Gran0(rn);
		A[i]+=T*(2.0*AM0*rn-1.0);
		rs[i]=rn;
	}
};
//---------------------------------------------------------------------------

#define FMAX 1000.0
#define b11 0.6
#define b12 0.12 
#define b22 0.1
#define a11 0.0 
#define a12 0.0 
#define a22 0.0 
#define slope11 2.5
#define slope22 2.5
#define slope12 (slope11+slope22)/2.0
#define constforca11 slope11*0.4
#define constforca12 slope12*0.4
#define constforca22 slope22*0.4

__global__ void G_update(int N,part *Gp,REAL alpha,REAL beta, REAL eta,REAL a2,REAL f0,REAL ra2,REAL L,int *rs,int Nslow)
{
    int idx=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    int i,rn1,rn2;
    __shared__ REAL Sx[N_tot];
    __shared__ REAL Sy[N_tot];
    if(threadIdx.x==0)
     {	
	for(i=0;i<N_tot;i++)
	  {	
	     Sx[i]=Gp[i].x;
	     Sy[i]=Gp[i].y;
	  }
     }
    __syncthreads();
    REAL dx,dy,Lh,r,r2,Fx,Fy,b,Vx,Vy,Tx,Ty;
    if(idx<N_tot){
      Lh=L*0.5;
      Fx=0.0;
      Fy=0.0;
      Vx=0.0;
      Vy=0.0;
      for (i=0; i<N; i++) {
        if(i!=idx)
	{   // calculate distance taking PBC into account
	  dx=Sx[idx]-Sx[i];
	  dy=Sy[idx]-Sy[i];
	  // Periodic boundary conditions
	  if(dx>Lh)dx=dx-L;   else if(dx<-Lh)dx=L+dx;
	  if(dy>Lh)dy=dy-L;   else if(dy<-Lh)dy=L+dy;
	  r2=dx*dx+dy*dy;
	  //check if particles are interacting
	  if(r2<ra2)
	  //if(r2<0.3025)
	  {  
	    r=sqrt(r2);
	    if (r2<=a2) 
	    { //hard core replusion
	      b=FMAX/r;
	      Fx+=dx*b;
	      Fy+=dy*b;
	    }
	    else
	    { //valid force range
	       if(idx<Nslow&&i<Nslow)
	       {
	       Fx+=b11*dx*((constforca11/r)-slope11);
	       Fy+=b11*dy*((constforca11/r)-slope11);
	       }
	       else
	       {
	         if(idx>=Nslow&&i>=Nslow)
	         {
		    Fx+=b22*dx*((constforca22/r)-slope22);
		    Fy+=b22*dy*((constforca22/r)-slope22);
                 }
		 else
		 {
		   if((idx>=Nslow&&i<Nslow)||(idx<Nslow&&i>=Nslow))
	           {
		     Fx+=b12*dx*((constforca12/r)-slope12);
		     Fy+=b12*dy*((constforca12/r)-slope12);
		   }
		 }		                                     
	      }
	   }      
	   //sum of neighbor velocities
	   if(idx<Nslow&&i<Nslow)
	   {
	      Vx+=Gp[i].vx*a11;
              Vy+=Gp[i].vy*a11;
	   }
	   if(idx>=Nslow&&i>=Nslow)
	   {
	      Vx+=Gp[i].vx*a22;
	      Vy+=Gp[i].vy*a22;
	   }
	   if((idx>=Nslow&&i<Nslow)||(idx<Nslow&&i>=Nslow))
	   {
	      Vx+=Gp[i].vx*a12;
	      Vy+=Gp[i].vy*a12;
	   }   
	   }
	}

   }
      //random angle
      rn1=Gran0(rs[idx]);
      rs[idx]=rn1;
      b=TWOPI*(AM0*rn1)-PI;
      rn2=Gran0(rs[idx]);
      rs[idx]=rn2;
      Tx=Vx+Fx+eta*cos(b);
      Ty=Vy+Fy+eta*sin(b);
      Gp[idx].theta=atan2(Ty,Tx);      
    } 
};


__global__ void G_timestep(int N,part *Gp,REAL dt,REAL L) {
//Bem melhor sem memoria compartilhada
   int idx=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
   REAL vx,vy,theta,v0,x,y;   
   if (idx<N) {
      //update velocity
      theta=Gp[idx].theta;
      v0=Gp[idx].v0;
      vx=v0*cos(theta);
      vy=v0*sin(theta);
      Gp[idx].vx=vx;
      Gp[idx].vy=vy;
      //update coordinate
      x=Gp[idx].x;
      y=Gp[idx].y;
      x+=dt*vx;while(x<0) x+=L;while(x>=L) x-=L;
      y+=dt*vy;while(y<0) y+=L;while(y>=L) y-=L;
      Gp[idx].x=x;
      Gp[idx].y=y;
      }
};

__global__ void G_gama(int N,part *Gp, REAL ra2,REAL Nslow)
{
__shared__ int Sna[N_tot];
__shared__ int Snv[N_tot];   
int idx=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
REAL dx,dy,r2;
int i;
if(idx<N){
   Sna[idx]=0;
   Snv[idx]=0;
}
if(idx<N){
   for(i=0;i<N;i++){
      if(i!=idx){
         dx=Gp[idx].x-Gp[i].x;
	 dy=Gp[idx].y-Gp[i].y;
	 r2=dx*dx+dy*dy;
	 //if(r2<ra2){
	 if(r2<0.3025){
	    if(i<Nslow)Snv[idx]++;
	    else Sna[idx]++;
	 }
      }
   }
}
//__syncthreads();
if(idx<N)
     {	
	Gp[idx].na=Sna[idx];
	Gp[idx].nv=Snv[idx];
     }
};

//---------------------------------------------------------------------------
//---------------------------------------------------------------------------
//-------parametros----------------------------------------------------------
particles::particles()
{  
    idum=2354;
    iyy=0;
    ir=new int[100];  iff=0;
    simtype=0;
    N=N_tot;   //total number
    Nslow=3000; //number of slow particles
    Nt=100000000; //number of timesteps
    Nto=1000; //output interval
    dt=1.0;
    vslow=0.007; //slow velocity
    vfast=0.007; //fast velocity
    alpha=0.0;
    beta=0.55;
    eta=1.0;
//    L=(2.*ceil(pow(N,0.5)*0.17))+4.; //system size
    L=19;
    a=0.2;  //particle size
    ra=0.55; //force radius
    f0=2.5; //force slope
    rs=123456; //random seed
    testecontinua=0;
    Gdev=0;
    outfn="part";
    Gp=NULL;
    GRs=NULL;
    p=NULL;
};


particles::~particles()
{
    if(GRs!=NULL) hipFree(GRs);
    if(Gp!=NULL) hipFree(Gp);
    if(p!=NULL) delete[] p;
    delete[] ir;
};

//---------------------------------------------------------------------------
//---------------------------------------------------------------------------

double particles::ran2(int iseed)
{
	//       based on ran2 from Numerical Recipes page 197 (fortran)
	//       meant to emulate the unix function rand
	//       positive iseed reinitializes, zero iseed - same string
	// ir - class-array int[100], ma,ia,ic,rm - constants
	// idum,iff,iyy - class-var (iff=0 before first run)
	double rand;
	int j;
	if((iseed>0) || (iff==0))
	{iff=1;
		idum=-iseed-3;
		idum=(ic-idum)%ma;       //-> idum < ma
		for(j=0;j<97;j++)  {
			idum=(ia*idum+ic)%ma;      //-> idum < ma
			ir[j]=idum;
		}
		idum=(ia*idum+ic)%ma;         //-> idum < ma
		iyy=idum;                     //-> iyy <ma
	}
	j=(97*iyy)/ma;                  //-> i < 97
	iyy=ir[j];
	rand=iyy*rm;
	idum=(ia*idum+ic)%ma;
	ir[j]=idum;
	if(rand==0.0) rand=1e-14; //double !!
	return rand;
}

//---------------------------------------------------------------------------
//---------------------------------------------------------------------------

void particles::simulate()
{
    int frame,i;
    double t;
    long int tnf=1,n,tempocontinuacao;
    REAL gama=0;
    FILE *arq1;
    FILE *arq2;
    FILE *arq3;
    frame=0;
    t=0.0;
//    abrindo o arquivo com informacoes do estado final do sistema interrompido
//  tudo ainda em cpu... 
   if(testecontinua==1)
     {
	arq3=fopen("estado-final","r");
	fscanf(arq3,"%d %d %li %d %d %f %f %f %f %f %f",&N,&Nslow,&n,&tnf,&frame,&alpha,&beta,&eta,&a,&ra,&L);	
	printf("%d %d %d %li %d %f %f %f %f %f %f\n",N,Nslow,n,tnf,frame,alpha,beta,eta,a,ra,L);
	//alocando vetor de p na CPU
	p=new part[N];
	//alocando Gp na GPU
	memP=N*sizeof(part);
	hipMalloc((void**)&Gp,memP);
	printf("%f %f %f\n",f0,vslow,vfast);
	printf("N= %d, Nslow=%d\n",N,Nslow);
	for(i=0;i<N;i++)
	  {
	     fscanf(arq3,"%f %f %f %f",&p[i].x,&p[i].y,&p[i].vx,&p[i].vy);
	  }
	for(i=0;i<Nslow;i++)p[i].v0=vslow;
	for(i=Nslow;i<N;i++)p[i].v0=vfast;
	t=n*dt;
	arq2=fopen("posicoes.dat","a");
	// copia os dados de p* para a GPU
	hipMemcpy(Gp,p,memP,hipMemcpyHostToDevice);//(destino,fonte,memoria,hipMemcpyHostToDevice ou hipMemcpyDeviceToHost)	
	tempocontinuacao=n;
     }
   else
     {	
	arq1=fopen("dados.dat","w");
	fclose(arq1);
	arq2=fopen("posicoes.dat","w");
	fclose(arq2);
	output(frame,gama,t); //intial state, data is still on CPU
	tempocontinuacao=1;
     }
//    printf("alfa %f, beta %f\n",alpha,beta);
    for(n=tempocontinuacao;n<=Nt;n++)
    {        
        G_update<<<GRID,BLOCK>>>(N,Gp,alpha,beta,eta,a,f0,ra,L,GRs,Nslow); //update particle velocities
        //hipMemcpy(p,Gp,memP,hipMemcpyDeviceToHost); //copia da GPU para CPU 
        G_timestep<<<GRID,BLOCK>>>(N,Gp,dt,L); //integrate equation of motion
        t+=dt;
	if((n==1)||(n%200)==0)
        {	   
	   arq1=fopen("dados.dat","a");    
	   G_gama<<<GRID,BLOCK>>>(N,Gp,ra,Nslow); //calculo dos numeros de vizinhos na e nv
	   hipMemcpy(p,Gp,memP,hipMemcpyDeviceToHost); //copia da GPU para CPU 
	   gama=0;
	   for(i=0;i<Nslow;i++){if((p[i].na+p[i].nv)!=0)gama=gama+(p[i].na/(p[i].na+p[i].nv));}
	   gama/=Nslow;
	   outputgama(gama,n,arq1);
	   printf("n=%d, gama=%f\n",n,gama);
	   fflush(stdout);
	   fclose(arq1);
	   if((n%Nto==0)||(n==1))
	     {
		arq3=fopen("estado-final","w");
		outputfinal(N,Nslow,n,tnf,frame,alpha,beta,eta,a,ra,L,arq3);
		fclose(arq3);
	     }
        }	
	if(n>=tnf){
	   tnf=tnf+pow(tnf,0.8);
	   hipMemcpy(p,Gp,memP,hipMemcpyDeviceToHost);
	   arq2=fopen("posicoes.dat","a");
	   outputposition(n,arq2);
	   fclose(arq2);
	   fflush(stdout);
	}							
        if((n%Nto)==0)
        {
	    frame++;
            hipMemcpy(p,Gp,memP,hipMemcpyDeviceToHost);
	    output(frame,gama,t);
        }
    }
//fclose(arq1);
};

//---------------------------------------------------------------------------
int particles::output(int fno, REAL gama, REAL t)
{
//    int i;
//    fHandle f;
//    string s;

//    f=FileCreate(outfn+IntToStr(fno)+".dat");
//    s="#t= "+FloatToStr(t)+", N= "+IntToStr(N)+", L= "+FloatToStr(L)+", gama= "+FloatToStr(gama)+" a= "+FloatToStr(a)+"\n";
//    FileWrite(f,s.c_str(),s.length());
    
//    for(i=0;i<N;i++)
//    {
//        s=IntToStr(i+1)+"\t"+FloatToStr(p[i].x)+"\t"+FloatToStr(p[i].y)+"\t"+FloatToStr(p[i].v0)+"\t"+FloatToStr(p[i].theta)+"\n";
//    	FileWrite(f,s.c_str(),s.length());
//    }
    
//    FileClose(f);
    return 0;
};

//--------------------------------------------------------------------------
int particles::outputgama(REAL gama, int passo, FILE *arq1)
{
    fprintf(arq1,"%d %f\n",passo,gama); fflush(stdout);
    return 0;
};

//--------------------------------------------------------------------------
int particles::outputfinal(int N,int Nslow,long int n,long int tnf, int frame, float alpha, float beta, float eta, float a, float ra, float L, FILE *arq3)
{
   int i;
   // imprimindo parametros do sistema
   // write(30,*)l,tnf,q,a,raz
   // write(30,"(4(f10.6,1x))")x,y
   //printf("escreveu estado-final\n");
   fprintf(arq3,"%d %d %li %li %d %f %f %f %f %f %f \n",N,Nslow ,n,tnf,frame,alpha, beta,eta,a,ra,L); 
   //escreve posicoes e angulo
   for(i=0;i<N;i++)fprintf(arq3,"%f %f %f %f\n",p[i].x,p[i].y,p[i].vx,p[i].vy);
   fflush(stdout);
   return 0;
};


//--------------------------------------------------------------------------
int particles::outputposition(int n,FILE *arq2)
{
    int i;
    fprintf(arq2,"%d\n",n);
    for(i=0;i<Nslow;i++)fprintf(arq2,"%f %f %f %f\n",p[i].x,p[i].y,p[i].vx,p[i].vy);
    fprintf(arq2,"inicio ciano\n");
    for(i=Nslow;i<N;i++)fprintf(arq2,"%f %f %f %f\n",p[i].x,p[i].y,p[i].vx,p[i].vy);
    return 0;
}
			
//---------------------------------------------------------------------------
void particles::initparams(int n, char* argv[])
{//overwrite deault built-in parameters set in constructor
	int m; 
	paramfilereader *pf=new paramfilereader();
	m=0;
	if(n>1) m=pf->cmdlineopenread(n,argv);
	if(m<1) m=pf->openread("part.ini");
	if(m>0)
	{  
	   simtype=pf->getint("simtype",simtype);
	   Gdev=pf->getint("Gdev",Gdev);
	   testecontinua=pf->getint("c",testecontinua);
	   N=pf->getint("N",N);
	   Nslow=pf->getint("Nslow",Nslow);
	   Nt=pf->getint("Nt",Nt);
	   Nto=pf->getint("Nto",Nto);
	   dt=pf->getdouble("dt",dt);
	   L=pf->getdouble("L",L);
	   a=pf->getdouble("a",a);
	   alpha=pf->getdouble("alpha",alpha);
	   beta=pf->getdouble("beta",beta);
	   eta=pf->getdouble("eta",eta);
	   outfn=pf->getstring("output");
	   if(outfn=="") outfn="part";		

	}
	delete pf;
	
};

//---------------------------------------------------------------------------

int particles::initialize()
{
    double x;
    REAL angle;//,angulo, raio;
//    REAL raiosorteio;
    int i,j,k;
    hipError_t err;
    size_t fmem,tmem;
    int *iarr;

    hipDeviceReset();
    hipSetDevice(Gdev);
    hipMemGetInfo(&fmem,&tmem);
    printf("GPU memory before allocation free: %u, total: %u\n",fmem,tmem);
		
    BLOCK=dim3(MAXT,1);
    // blocs 512, 1024 , tentando 8000
    i=8; //x blocks, limited to 256^2 !!! because of the GPU and the cuda version
    //testando: i=8000 antes de mexer hoje (13/03/2014)
    k=i*BLOCK.x;
    j=(N+k-1)/k; //y blocks
    GRID=dim3(i,j);
    
    i=MAXT;
    printf("GPU execution layout:\n - threads: %d\n - system block: %dx%d\n - state block %d\n",i,GRID.x,GRID.y,BLOCK.x);
	
    memP=N*sizeof(part);
    memRs=N*sizeof(int);

    hipMalloc((void**)&Gp,memP);
    hipMalloc((void**)&GRs,memRs);
	
    ran2(rs);	
    iarr=new int[N];
	 
    for(i=0;i<N;i++) 
    { 
        x=ran2()*0xFFFFFFF+127983.0;
    	j=((int) x)^MASK0;
    	iarr[i]= j;
    }
    hipMemcpy(GRs, iarr,memRs,hipMemcpyHostToDevice);
    delete[] iarr;
	
    err=hipGetLastError();
    if(err!=hipSuccess)                                 
    printf("CUDA error [%d] (alloc) : %s\n",err,hipGetErrorString(err));
	
    hipMemGetInfo(&fmem,&tmem);
    printf("GPU memory after allocation free: %u, total: %u\n",fmem,tmem);
    p=new part[N];	
    if(testecontinua==0)
      {   	
	//sorteio inicial    
	for(i=0;i<N;i++)
	  {
	     if(i<Nslow) p[i].v0=vslow;
	     else p[i].v0=vfast;
	   //  raiosorteio=(L-5.)/2.;
	     // sorteio inicial redondo
	    // angulo=ran2()*TWOPI;
	    // raio=pow(ran2(),0.5);
	    // p[i].x=L/2.+raiosorteio*raio*cos(angulo);
	    // p[i].y=L/2.+raiosorteio*raio*sin(angulo);
	    p[i].x=L*ran2();
	    p[i].y=L*ran2();
	     // sorteio inicial das velocidades	
	     angle=TWOPI*ran2();
	     p[i].theta=angle;
	     p[i].vx=p[i].v0*cos(angle);
	     p[i].vy=p[i].v0*sin(angle);
	  }
	hipMemcpy(Gp,p,memP,hipMemcpyHostToDevice);  
     }   
    //since we need the particle size and force radius only in squared form we do:
    a*=a;
    ra*=ra;
    return 0;
};

//---------------------------------------------------------------------------
//---------------------------------------------------------------------------
int main(int argc, char* argv[])
{
	particles *pc=new particles();
	pc->initparams(argc,argv);
	printf("params set, initializing ... ");
	if(pc->initialize()!=0)
	{
	   printf("program stopped\n");
	   return 0;
	}
	printf("done\nstarting the simulation\n");
	if(pc->simtype==0) pc->simulate();
	printf("finished - cleaning up\n");
	delete pc;
	return 0;
}
//---------------------------------------------------------------------------



