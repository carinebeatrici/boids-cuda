#include "hip/hip_runtime.h"
#include <time.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
//#include <unistd.h>
#include <hip/hip_runtime.h>
//#include <hipfft/hipfft.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "param.h"
#include "particle.h"

#define TWO_PI 6.283185307f


__global__ void G_update_position(particle *G_boid, float
*G_Fx, float *G_Fy ,float *G_sum_Vx, float *G_sum_Vy, hiprandState_t
*state, unsigned int * G_random_number)
{
   //Bem melhor sem memoria compartilhada
   int i=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
   //   float x,y;   
   float module_displace;
   float component_x,component_y;
   float angle;
   if (i<N) 
     {
        G_random_number[i] = hiprand(state) % RAND_MAX;
	angle = TWO_PI * (float) G_random_number[i] /RAND_MAX;
	component_x =  ( G_sum_Vx[i] + G_Fx[i] + ETA * cos(angle));
	component_y =  ( G_sum_Vy[i] + G_Fy[i] + ETA * sin(angle));
	module_displace = sqrt((component_x*component_x)+(component_y*component_y));
	G_boid[i].x = G_boid[i].x + G_boid[i].v0 * component_x/module_displace;
	G_boid[i].y = G_boid[i].y + G_boid[i].v0 * component_y/module_displace;
	//boundary conditions
	if(G_boid[i].x>=L_TENTATIVA)G_boid[i].x=G_boid[i].x-L_TENTATIVA;
	if(G_boid[i].y>=L_TENTATIVA)G_boid[i].y=G_boid[i].y-L_TENTATIVA;
	if(G_boid[i].x<0)G_boid[i].x=G_boid[i].x+L_TENTATIVA;
	if(G_boid[i].y<0)G_boid[i].y=G_boid[i].y+L_TENTATIVA;
	G_boid[i].vx = G_boid[i].v0 * component_x/module_displace;
	G_boid[i].vy = G_boid[i].v0 * component_y/module_displace; 
     }
};


void  update_position(particle *boid, float *fx, float *fy, float *sum_vx, float *sum_vy)
{
   float angle;
   int i;
   float module_displace;
   float componente_x,componente_y;
   for(i=0;i<N;i++)
     {
	angle = drand48() * TWO_PI;
	componente_x =  ( sum_vx[i] + fx[i] + ETA * cos(angle));
	componente_y =  ( sum_vy[i] + fy[i] + ETA * sin(angle));
	module_displace = sqrt((componente_x*componente_x)+(componente_y*componente_y));
	boid[i].x = boid[i].x + boid[i].v0 * componente_x/module_displace;
	boid[i].y = boid[i].y + boid[i].v0 * componente_y/module_displace;
	// condicoes de contorno
	if(boid[i].x>=L_TENTATIVA)boid[i].x=boid[i].x-L_TENTATIVA;
	if(boid[i].y>=L_TENTATIVA)boid[i].y=boid[i].y-L_TENTATIVA;
	if(boid[i].x<0)boid[i].x=boid[i].x+L_TENTATIVA;
	if(boid[i].y<0)boid[i].y=boid[i].y+L_TENTATIVA;
	boid[i].vx = boid[i].v0 * componente_x/module_displace;
	boid[i].vy = boid[i].v0 * componente_y/module_displace; 
     }   
}

